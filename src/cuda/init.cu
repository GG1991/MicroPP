
#include "micropp.hpp"
#include "common.hpp"

template<>
void micropp<3>::cuda_init()
{
	hipMalloc((void **)cuda_params.elem_type_d, nelem * sizeof(int));
	hipMemcpy(cuda_params->elem_type_d, elem_type, 
		   nelem * sizeof(int), hipMemcpyHostToDevice);
}

template<>
void micropp<3>::cuda_finalize()
{
	hipFree(cuda_params.elem_type_d);
}
