#include "hip/hip_runtime.h"
/*
 *  This source code is part of Micropp: a Finite Element library
 *  to solve composite materials micro-scale problems.
 *
 *  Copyright (C) - 2018
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */


#include "micropp.hpp"
#include "common.hpp"
#include "cuda.hpp"


__device__ material_t *material_list_d[MAX_MATERIALS];
material_base *material_base_list_d;
//struct cuda_params_t cuda_params;
int *elem_type_d;


__global__
void device_init_material(material_base *material_base_list_d)
{
	for (int i = 0; i < MAX_MATERIALS; ++i) {
		material_list_d[i] = material_t::make_material(material_base_list_d[i]);
	}
}

__global__
void device_delete_material()
{
	for (int i = 0; i < MAX_MATERIALS; ++i) {
		delete material_list_d;
	}
}

template<>
void micropp<3>::cuda_init(const micropp_params_t &params)
{
	hipMalloc((void **)&elem_type_d, nelem * sizeof(int));

	hipMalloc((void **)&material_base_list_d, MAX_MATERIALS * sizeof(material_base));
	for (int i = 0; i < MAX_MATERIALS; ++i) {
		hipMemcpy(&material_base_list_d[i], &params.materials[i], 
				sizeof(material_base), hipMemcpyHostToDevice);
	}

	device_init_material<<<1, 1>>>(material_base_list_d);

	hipMemcpy(elem_type_d, elem_type, 
		   nelem * sizeof(int), hipMemcpyHostToDevice);
}

template<>
void micropp<3>::cuda_finalize()
{
	//hipFree(cuda_params.elem_type_d);
}
