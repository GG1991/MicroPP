/*
 *  This source code is part of Micropp: a Finite Element library
 *  to solve composite materials micro-scale problems.
 *
 *  Copyright (C) - 2018
 *
 *  This program is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <https://www.gnu.org/licenses/>.
 */


#include "micropp.hpp"
#include "common.hpp"


template<>
void micropp<3>::cuda_init()
{
	hipMalloc((void **)&cuda_params.elem_type_d, nelem * sizeof(int));
	hipMemcpy(cuda_params.elem_type_d, elem_type, 
		   nelem * sizeof(int), hipMemcpyHostToDevice);
}

template<>
void micropp<3>::cuda_finalize()
{
	hipFree(cuda_params.elem_type_d);
}
